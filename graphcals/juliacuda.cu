#include "hip/hip_runtime.h"
#ifndef TDBK_JULIA
#define TDBK_JULIA

#include "julia.h"

__device__ Complex complexSquareJ(Complex z)
{
	Complex zSquare;

	zSquare.x = (z.x * z.x) - (z.y * z.y);
	zSquare.y = 2 * z.x * z.y;

	return zSquare;
}

__global__ void juliaSqTransform(int maxIter, WindowInfo windowInfo, Complex c, Complex *points, int *iterations)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (index < windowInfo.arrayLength)
    {
        int counter = 0;
	    Complex z;
	    
	    z = points[index];

	    while ((z.x * z.x + z.y * z.y <= 4.0) && (counter < maxIter))
	    {
		    z = complexSquareJ(z);
		    z.x += c.x;
		    z.y += c.y;
		    counter++;
	    }
	    
	    iterations[index] = counter;
    }
}

void julia(int nx, int ny, int maxIter, Complex c)
{
    Complex z, zIncr;
    int counter = 0;
    int arraySize = nx * ny;
    double realIterator = 0.0;
    double imaginaryIterator = 0.0;
    
    WindowInfo winInfo;
    winInfo.arrayLength = arraySize;
    winInfo.xComplexMin = xComplexMin;
    winInfo.xComplexMax = xComplexMax;
    winInfo.yComplexMin = yComplexMin;
    winInfo.yComplexMax = yComplexMax;
    
    int size = arraySize * sizeof(Complex);
    Complex *points = (Complex *)malloc(size);
    Complex *dev_points;
    hipMalloc((void **) &dev_points, size);
    
    int it_size = arraySize * sizeof(int);
    int *iterations = (int *)malloc(it_size);
    int *dev_iterations;
    hipMalloc((void **) &dev_iterations, it_size);
    
    int nThreads = 1024;
    int nBlocks = (arraySize + nThreads - 1) / nThreads;
    
    zIncr.x = complexWidth / double(nx);
    zIncr.y = complexHeight / double(ny);
    
    for (realIterator = xComplexMin; realIterator < xComplexMax - zIncr.x; realIterator += zIncr.x)
    {
        for (imaginaryIterator = yComplexMin; imaginaryIterator < yComplexMax - zIncr.y; imaginaryIterator += zIncr.y)
    	{
    	    z.x = realIterator;
    	    z.y = imaginaryIterator;
    	    
    	    points[counter] = z;
    	    
    	    counter++;
    	}
    }

    hipMemcpy(dev_points, points, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_iterations, iterations, it_size, hipMemcpyHostToDevice);
    
    juliaSqTransform<<< nBlocks, nThreads >>>(maxIter, winInfo, c, dev_points, dev_iterations);
    
    hipMemcpy(points, dev_points, size, hipMemcpyDeviceToHost);
    hipMemcpy(iterations, dev_iterations, it_size, hipMemcpyDeviceToHost);

    vector<Color> colorSet = GetCurrentColorSet();
    
    glBegin(GL_POINTS);
        for (int i = 0; i < arraySize; i++)
        {
            Complex point = points[i];
            int iterCount = iterations[i];
            
            SetColorAndPlot(maxIter, iterCount, point, colorSet);
        }
    glEnd();
    
    hipFree(dev_points);
    hipFree(dev_iterations);
    free(points);
    free(iterations);
}

#endif
