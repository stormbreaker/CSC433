#include "hip/hip_runtime.h"
#ifndef TDBK_MANDELBROT
#define TDBK_MANDELBROT

#include "calculations.h"
#include <stdio.h>
#include <chrono>

using namespace std;

/*
	Name: complexSquare
	Author: code from book modified by Benjamin Kaiser and Taylor Doell
	Description: This function squares the complex number. This function is tagged
    with __device__ to put the function on the card to allow the cuda card to
    easily call it.
*/
__device__ Complex complexSquare(Complex z)
{
	Complex zSquare;

    // Square the complex number
	zSquare.x = (z.x * z.x) - (z.y * z.y);
	zSquare.y = 2 * z.x * z.y;

	return zSquare;
}

/*
	Name: mandelSqTransform
	Author: code from book modified by Benjamin Kaiser and Taylor Doell
	Description: This function performs the calculations finding how fast the
    point diverges for the mandelbrot set. This function has the __global__ tag
    in order for the cuda card to be able to call this function.
*/
__global__ void mandelSqTransf(int maxIter, WindowInfo windowInfo,  Complex *points, int *iterations)
{
    // Get the current index from the thread and block information
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Make sure we are in the contents of the array of points so we don't
    // overstep the bounds
    if (index < windowInfo.arrayLength)
    {
	    int counter = 0;
        Complex z0;
	    Complex z;

        // Set initial points for calculations
	    z.x = 0;
	    z.y = 0;

        z0 = points[index];

        // Loop until point diverges or exceeds 4.0
	    while ((z.x * z.x + z.y * z.y <= 4.0) && (counter < maxIter))
	    {
            // Calculate the complex square and increment each point
		    z = complexSquare(z);
		    z.x += z0.x;
		    z.y += z0.y;
		    counter++; // Increment counter for iterations used
	    }

        // Store the iteration count
	    iterations[index] = counter;
    }
}

/*
	Name: juliaSqTransform
	Author: code from book modified by Benjamin Kaiser and Taylor Doell
	Description: This function uses the point from the mandelbrot set to help
    calculate the julia set and uses that to see how fast the point diverges.
    This function has the __global__ tag in order for the cuda card to be able
    to call this function.
*/
__global__ void juliaSqTransform(int maxIter, WindowInfo windowInfo, Complex c, Complex *points, int *iterations)
{
    // Get the current index from the thread and block information
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Make sure we are in the contents of the array of points so we don't
    // overstep the bounds
    if (index < windowInfo.arrayLength)
    {
        int counter = 0;
	    Complex z;

        // Set initial points for calculations
	    z = points[index];

        // Loop until point diverges or exceeds 4.0
	    while ((z.x * z.x + z.y * z.y <= 4.0) && (counter < maxIter))
	    {
		    z = complexSquare(z);
		    z.x += c.x;
		    z.y += c.y;
		    counter++;
	    }

        // Store the iteration count
	    iterations[index] = counter;
    }
}

/*
	Name: calculateSetParallel
	Author: Benjamin Kaiser and Taylor Doell
	Description: This is the main function that handles both the mandelbrot and
    the julia set in parallel. A bool flag tells the function which set to
    calculate. There is an optional parameter that can be used if the julia set
    is the set to calculate. Once the number (iterCount) is found, it does some
    calculations to divide the colors that were provided by the color set. It
    then plots the point.
*/
void calculateSetParallel(int nx, int ny, int maxIter, bool isMandelBrot, Complex c)
{
	Complex z, zIncr;
    int counter = 0;
    int arraySize = nx * ny;
    double realIterator = 0.0;
    double imaginaryIterator = 0.0;

    // Store the winInfo to help with set calculations
    WindowInfo winInfo;
    winInfo.arrayLength = arraySize;
    winInfo.xComplexMin = xComplexMin;
    winInfo.xComplexMax = xComplexMax;
    winInfo.yComplexMin = yComplexMin;
    winInfo.yComplexMax = yComplexMax;

    // Intialize size for the array of points
    int size = arraySize * sizeof(Complex);
    Complex *points = (Complex *)malloc(size);
    Complex *dev_points;
    // Allocate size on the card for the points
    hipMalloc((void **) &dev_points, size);

    // Intialize size for the array of iteration counts
    int it_size = arraySize * sizeof(int);
    int *iterations = (int *)malloc(it_size);
    int *dev_iterations;
    // Allocate size on the card for the values
    hipMalloc((void **) &dev_iterations, it_size);

    // Initilize thread and block count
    int nThreads = 1024;
    int nBlocks = (arraySize + nThreads - 1) / nThreads;

    // Calculate the increment values for the loop of points
    zIncr.x = complexWidth / double(nx);
    zIncr.y = complexHeight / double(ny);

    // Loop through to create the points for all the calculations
    for (realIterator = xComplexMin; realIterator < xComplexMax - zIncr.x; realIterator += zIncr.x)
    {
        for (imaginaryIterator = yComplexMin; imaginaryIterator < yComplexMax - zIncr.y; imaginaryIterator += zIncr.y)
    	{
            // Set real and imaginary parts of the point to be calculated
    	    z.x = realIterator;
    	    z.y = imaginaryIterator;

            // Store point
    	    points[counter] = z;

            // Increment counter
    	    counter++;
    	}
    }

    // Copy memory to the device
    hipMemcpy(dev_points, points, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_iterations, iterations, it_size, hipMemcpyHostToDevice);

    // Calculate currently selected set on the cuda card
    if (isMandelBrot == true)
    {
        mandelSqTransf<<< nBlocks, nThreads >>>(maxIter, winInfo, dev_points, dev_iterations);
    }
    else
    {
        juliaSqTransform<<< nBlocks, nThreads >>>(maxIter, winInfo, c, dev_points, dev_iterations);
    }

	auto start = chrono::system_clock::now();

    // Copy memory back to get the values
    hipMemcpy(points, dev_points, size, hipMemcpyDeviceToHost);
    hipMemcpy(iterations, dev_iterations, it_size, hipMemcpyDeviceToHost);

    // Get the current color set to draw the pixels with
    vector<Color> colorSet = GetCurrentColorSet();

	chrono::duration<double> test = chrono::system_clock::now() - start;


	//uncomment the following line to see timing output
	//cout << "Parallel: " << test.count() << " seconds" << endl;

    // Loop through all the points and plot them in the window
    glBegin(GL_POINTS);
        for (int i = 0; i < arraySize; i++)
        {
            Complex point = points[i];
            int iterCount = iterations[i];

            SetColorAndPlot(maxIter, iterCount, point, colorSet);
        }
    glEnd();

    // Free up all the memory
    hipFree(dev_points);
    hipFree(dev_iterations);
    free(points);
    free(iterations);
}

#endif
